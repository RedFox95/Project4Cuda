#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <string>

using namespace std;

struct fileinfo {
    int numLines;
    int lineLength;
};

struct matchLocation {
    int x; // row 
    int y; // col
    int pl; // pattern line
};

struct matchCoordinate {
    int x;
    int y;
};


 //Get the fileinfo struct for this file, containing the number of lines and length of a line.

fileinfo getFileInfo(string filename) {
    ifstream lineCounter(filename);
    int numLines = 0;
    string line;
    while (getline(lineCounter, line)) numLines++;
    lineCounter.close();
    struct fileinfo retVal = { numLines, line.length() };
    return retVal;
}


 //Returns the upper leftmost coordinate of a full match, otherwise return null if no full match.

matchCoordinate searchForRealMatches(matchLocation match, matchLocation** allMatches, int* numMatchesArr, int numPatternLines, int world_size) {
    //cout << "-> searchForRealMAtches x:" << match.x << " y: " << match.y << " pl: " << match.pl << endl;
    matchLocation** patternMatchLocations = new matchLocation * [numPatternLines];
    for (int i = 0; i < numPatternLines; i++) patternMatchLocations[i] = nullptr;

    patternMatchLocations[match.pl] = &match;
    for (int i = 0; i < world_size; i++) {
        for (int j = 0; j < numMatchesArr[i]; j++) {
            bool fullMatch = true;
            // for each match...
            if (allMatches[i][j].y == match.y && allMatches[i][j].x != match.x && allMatches[i][j].pl != match.pl) {
                // if it's in the correct column
                for (int k = 0; k < numPatternLines; k++) {
                    if (allMatches[i][j].x == match.x + k && allMatches[i][j].pl == match.pl + k) {
                        // this is a corresponding match!
                        patternMatchLocations[allMatches[i][j].pl] = &allMatches[i][j];
                    }
                    // check if full match 
                    if (patternMatchLocations[k] == nullptr) fullMatch = false;
                }
                if (fullMatch) {
                    struct matchCoordinate retVal = { patternMatchLocations[0]->x, patternMatchLocations[0]->y };
                    delete[] patternMatchLocations;
                    return retVal;
                }
            }
        }
    }
    // return -1, -1 if no match found 
    struct matchCoordinate retVal = { -1, -1 };
    return retVal;
}

__global__ void findPartialMatches(char**inputLines, char**patternLines, int*numInputLines, int*lenInputLines, int*numPatternLines, int*lenPatternLines, int*numMatchesArr, matchLocation**allMatches, int*numThreads) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int totalNumThreads = blockDim.x * (*numThreads);
    numMatchesArr[threadId] = 0; // initialize all numMatches to 0
    if (threadId <= numInputLines) {
        int sizeOfMatchArr = 10; // 10 for now... then dynamically increase if needed
        allMatches[threadId] = new matchLocation[sizeOfMatchArr]; // store the matches at the index for this thread
        for (int j = 0; j < numPatternLines; j++) {
            int pos = 0;
            string jPatternLine(patternLines[j], patternLines[j] + lenPatternLines-1);
            string iInputLine(inputLines[threadId], inputLines[threadId] + lenInputLines-1);
            int found = iInputLine.find(jPatternLine, pos);
            while (found != string::npos) {
                if (numMatchesArr[threadId] >= sizeOfMatchArr) {
                    // increase matchArr size 
                    int biggerSize = sizeOfMatchArr * 2;
                    matchLocation* biggerArr = new matchLocation[biggerSize];
                    memcpy(biggerArr, matchArr, sizeof(matchLocation) * numMatchesArr[threadId]);
                    delete[] matchArr;
                    matchArr = biggerArr;
                    sizeOfMatchArr = biggerSize;
                }
                // store the match
                struct matchLocation m = { threadId, found, j }; // where threadId is the row number, found is the col number, and j is the pattern line number
                matchArr[numMatchesArr[threadId]] = m;
                // update pos, numMatches, and found for the next iteration
                pos = found + 1;
                numMatchesArr[threadId]++; // TODO use atomic cuda increment
                found = iInputLine.find(jPatternLine, pos);
            }
        }
    }
}

int main(int argc, char** argv) {
    // get info about input file
    string inputFile = argv[1];
    fileinfo inputInfo = getFileInfo(inputFile);
    int numInputLines = inputInfo.numLines;
    int lenInputLines = inputInfo.lineLength + 1;

    // read the input file in line by line and store in array
    ifstream file(inputFile);
    char** inputLines = new char* [numInputLines]; // num rows (lines)
    for (int i = 0; i < numInputLines; i++) {
        inputLines[i] = new char[lenInputLines]; // num cols (line length)
    }

    string line;
    int lineNum = 0; // for indexing into the allLines arr
    while (getline(file, line)) {
        strcpy_s(inputLines[lineNum], lenInputLines, line.c_str());
        lineNum++;
    }

    // get info about the pattern file
    string patternFile = argv[2];
    fileinfo patternInfo = getFileInfo(patternFile);
    int numPatternLines = patternInfo.numLines;
    int lenPatternLines = patternInfo.lineLength + 1;

    // read the pattern file in line by line and store in array
    ifstream patternFileStream(patternFile);
    char** patternLines = new char* [numPatternLines]; // num rows (lines)
    for (int i = 0; i < numPatternLines; i++) {
        patternLines[i] = new char[lenPatternLines]; // num cols (line length)
    }
    lineNum = 0; // for indexing into the pattern arr
    while (getline(patternFileStream, line)) {
        strcpy_s(patternLines[lineNum], lenPatternLines, line.c_str());
        lineNum++;
    }

    // allocate memory on device for inputLines and patternLines and copy to the device memory
    char ** inputLinesDevice;
    hipMalloc(&inputLinesDevice, numInputLines * sizeof(char*)); 
    hipMemcpy(inputLinesDevice, inputLines, numInputLines * sizeof(char*), hipMemcpyHostToDevice);
    for (int i = 0; i < numInputLines; i++) {
        hipMalloc(&inputLinesDevice[i], lenInputLines * sizeof(char));
        hipMemcpy(inputLinesDevice[i], inputLines[i], lenInputLines * sizeof(char), hipMemcpyHostToDevice);
    }
    char ** patternLinesDevice;
    hipMalloc(&patternLinesDevice, numPatternLines * sizeof(char*)); 
    hipMemcpy(patternLinesDevice, patternLines, numPatternLines * sizeof(char*), hipMemcpyHostToDevice);
    for (int i = 0; i < numPatternLines; i++) {
        hipMalloc(&patternLinesDevice[i], lenPatternLines * sizeof(char));
        hipMemcpy(patternLinesDevice[i], patternLines[i], lenPatternLines * sizeof(char), hipMemcpyHostToDevice);
    }

    // set the number of blocks and number of threads we want to use
    int numBlocks = 1;
    int numThreads = 32;
    // guessing how to compute this... - maybe for smaller test files we can change this around but for larger it needs to be set
    // if (numInputLines <= 1024) {
    //     numThreads = numInputLines; // according to the internet, 1024 is the max number of threads in a block
    // } else {
    //     numBlocks = ; // ???
    //     numThreads = ; // ???
    // }
    int totalNumThreads = numBlocks * numThreads;

    // setup pointers to get the results from device memory in allMatchLocations and numMatchesArr
    matchLocation** allMatchLocationsDevice;
    hipMalloc(&allMatchLocationsDevice, numBlocks * sizeof(matchLocation*)); 

    int* numMatchesArrDevice;
    hipMalloc(&numMatchesArrDevice, numBlocks * sizeof(int)); 
    for (int i = 0; i < numBlocks; i++) {
        hipMalloc(&numMatchesArrDevice[i], numBlocks * sizeof(int)); 
    }

    // start the kernel to find partial matches
    findPartialMatches<<<numBlocks,numThreads>>>(inputLinesDevice, patternLinesDevice, &numInputLines, &lenInputLines, &numPatternLines, &lenPatternLines, allMatchLocationsDevice, numMatchesArrDevice, &numThreads);

    // copy the results to host memory
    int* numMatchesArr = new int[totalNumThreads];
    matchLocation** allMatchLocations = new matchLocation * [totalNumThreads];

    for (int i = 0; i < totalNumThreads; i++) {
        hipMemcpy(numMatchesArrDevice[i], numMatchesArr[i], sizeof(int), hipMemcpyDeviceToHost);
        if (numMatchesArr[i] > 0) {
            hipMemcpy(allMatchLocationsDevice[i], allMatchLocations[i], numMatchesArr[i] * (matchLocation*), hipMemcpyDeviceToHost);
        }
    }
    

    // prep the output file 
    ofstream outputFile("output.txt");

    // compare all the matches for lines in a pattern file to find full matches
    int sizeOfCoordArr = 10; // 10 for now... then dynamically increase if needed
    matchCoordinate* coordArr = new matchCoordinate[sizeOfCoordArr];
    int numCoords = 0;
    for (int i = 0; i < numBlocks; i++) {
        for (int j = 0; j < numMatchesArr[i]; j++) {
            matchCoordinate coor = searchForRealMatches(allMatchLocations[i][j], allMatchLocations, numMatchesArr, numPatternLines, world_size);
            if (coor.x == -1 && coor.y == -1) continue; // not a match
            bool alreadyFound = false;
            for (int k = 0; k < numCoords; k++) {
                if (coordArr[k].x == coor.x && coordArr[k].y == coor.y) alreadyFound = true;
            }
            if (alreadyFound) continue; // go to next match
            if (numCoords >= sizeOfCoordArr) {
                // increase coordArr size 
                int biggerSize = sizeOfCoordArr * 2;
                matchCoordinate* biggerArr = new matchCoordinate[biggerSize];
                memcpy(biggerArr, coordArr, sizeof(matchCoordinate) * numCoords);
                delete[] coordArr;
                coordArr = biggerArr;
                sizeOfCoordArr = biggerSize;
            }
            coordArr[numCoords] = coor;
            numCoords++;
            cout << "MATCH AT: " << coor.x << ", " << coor.y << endl;
            // output as column, row and ensure the coordinates are not 0 indexed
            outputFile << coor.y + 1 << ", " << coor.x + 1 << "\n";
        }
    }
    outputFile.close();
    // cleanup memory 
    for (int i = 0; i < numInputLines; i++) {
        hipFree(inputLinesDevice[i]);
        delete[] inputLines[i];
    }
    delete[] inputLines;
    hipFree(inputLinesDevice);
    for (int i = 0; i < numPatternLines; i++) {
        hipFree(patternLinesDevice[i]);
        delete[] patternLines[i];
    }
    delete[] patternLines;
    hipFree(patternLinesDevice);
    for (int i = 0; i < totalNumThreads; i++) {
        hipFree(allMatchLocationsDevice[i]);
        delete[] allMatchLocations[i];
    }
    delete[] allMatchLocations;
    hipFree(allMatchLocationsDevice);
    delete[] numMatchesArr;
    hipFree(numMatchesArrDevice);
    delete[] coordArr;

    return 0;
}
