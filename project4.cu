#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <string>

using namespace std;

struct fileinfo {
    int numLines;
    size_t lineLength;
};

struct matchLocation {
    int x; // row 
    int y; // col
    int pl; // pattern line
};

struct matchCoordinate {
    int x;
    int y;
};


//Get the fileinfo struct for this file, containing the number of lines and length of a line.

fileinfo getFileInfo(string filename) {
    ifstream lineCounter(filename);
    int numLines = 0;
    string line;
    while (getline(lineCounter, line)) numLines++;
    lineCounter.close();
    struct fileinfo retVal = { numLines, line.length() };
    return retVal;
}


//Returns the upper leftmost coordinate of a full match, otherwise return null if no full match.

matchCoordinate searchForRealMatches(matchLocation match, matchLocation* allMatches, int numMatches, int numPatternLines, int totalNumThreads) {
    //cout << "-> searchForRealMAtches x:" << match.x << " y: " << match.y << " pl: " << match.pl << endl;
    matchLocation** patternMatchLocations = new matchLocation * [numPatternLines];
    for (int i = 0; i < numPatternLines; i++) patternMatchLocations[i] = nullptr;

    patternMatchLocations[match.pl] = &match;
    for (int i = 0; i < numMatches; i++) {
        bool fullMatch = true;
        // for each match...
        if (allMatches[i].y == match.y && allMatches[i].x != match.x && allMatches[i].pl != match.pl) {
            // if it's in the correct column
            for (int k = 0; k < numPatternLines; k++) {
                if (allMatches[i].x == match.x + k && allMatches[i].pl == match.pl + k) {
                    // this is a corresponding match!
                    patternMatchLocations[allMatches[i].pl] = &allMatches[i];
                }
                // check if full match 
                if (patternMatchLocations[k] == nullptr) fullMatch = false;
            }
            if (fullMatch) {
                struct matchCoordinate retVal = { patternMatchLocations[0]->x, patternMatchLocations[0]->y };
                delete[] patternMatchLocations;
                return retVal;
            }
        }
       
    }
    // return -1, -1 if no match found 
    struct matchCoordinate retVal = { -1, -1 };
    return retVal;
}

__device__ void findSubStr(char*str, int row, int strLen, char* subStr, int patternLineNum, int subStrLen, int pos, int*foundPos) {
    printf("-> findSubStr row %d, strLen %d, patternLineNum %d, subStrLen %d, pos %d\n", row, strLen, patternLineNum, subStrLen, pos);
    *foundPos = -1; // assume no match found
    for (int i = pos; i <= strLen - subStrLen; i++) {
        //printf("r %d: i is %d\n", row, i);
        bool found = true;
        for (int j = 0; j < subStrLen; j++) {
            //printf("r %d: j is %d and i+j is %d comparing %c vs %c\n", row, j, i + j, str[(row * strLen) + i + j], subStr[(patternLineNum * subStrLen)  + j]);
            if (str[(row*strLen) + i + j] != subStr[(patternLineNum * subStrLen)+j]) {
                found = false;
                break;
            }
        }
        if (found) {
            *foundPos = i;
            //printf("r %d: FOUND match in device foundPos is %d\n", row, *foundPos);
            break;
        }
    }
}

__global__ void findPartialMatches(char* inputLines, char* patternLines, int* numInputLines, int* lenInputLines, int* numPatternLines, int* lenPatternLines, int* numMatches, matchLocation* allMatches) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndex = threadId * *lenInputLines;
    //printf("Tmg thread id is %d and index is %d\n", threadId, threadIndex);
    if (threadId < *numInputLines) {
        for (int j = 0; j < *numPatternLines; j++) {
            int pos = 0;
            int found = -1;
            findSubStr(inputLines, threadId, *lenInputLines, patternLines, j, *lenPatternLines, 0, &found);
            while (found != -1) {
                // store the match
                struct matchLocation m = { threadId, found, j }; // where threadId is the row number, found is the col number, and j is the pattern line number
                int storingIndex = atomicAdd(numMatches, 1);
                allMatches[storingIndex] = m;
                // update pos, numMatches, and found for the next iteration
                pos = found + 1;
                findSubStr(inputLines, threadId,  *lenInputLines, patternLines, j, *lenPatternLines, pos, &found);
            }
        }
    }
}

int main(int argc, char** argv) {
    // get info about input file
    string inputFile = argv[1];
    fileinfo inputInfo = getFileInfo(inputFile);
    int numInputLines = inputInfo.numLines;
    int lenInputLines = inputInfo.lineLength;

    // read the input file in line by line and store in array
    ifstream file(inputFile);
    char** inputLines = new char* [numInputLines]; // num rows (lines)
    for (int i = 0; i < numInputLines; i++) {
        inputLines[i] = new char[lenInputLines+1]; // num cols (line length)
    }

    // temp test..
    char* inputLine1d = new char[numInputLines * lenInputLines];

    string line;
    int lineNum = 0; // for indexing into the allLines arr
    while (getline(file, line)) {
        strcpy_s(inputLines[lineNum], lenInputLines+1, line.c_str());
        for (int i = 0; i < lenInputLines; i++) {
            inputLine1d[(lineNum * lenInputLines) + i] = line.c_str()[i];
        }  
        lineNum++;
    }
    // get info about the pattern file
    string patternFile = argv[2];
    fileinfo patternInfo = getFileInfo(patternFile);
    int numPatternLines = patternInfo.numLines;
    int lenPatternLines = patternInfo.lineLength;

    // read the pattern file in line by line and store in array
    ifstream patternFileStream(patternFile);
    char** patternLines = new char* [numPatternLines]; // num rows (lines)
    for (int i = 0; i < numPatternLines; i++) {
        patternLines[i] = new char[lenPatternLines+1]; // num cols (line length)
    }
    char* patternLine1d = new char[numPatternLines*lenPatternLines];

    lineNum = 0; // for indexing into the pattern arr
    while (getline(patternFileStream, line)) {
        strcpy_s(patternLines[lineNum], lenPatternLines+1, line.c_str());
        for (int i = 0; i < lenPatternLines; i++) {
            patternLine1d[(lineNum * lenPatternLines) + i] = line.c_str()[i];
        }
        lineNum++;
    }
    // allocate memory on device for inputLines and patternLines and copy to the device memory
    char* inputLinesDevice;
    hipMalloc((void**)&inputLinesDevice, lenInputLines * numInputLines * sizeof(char)); // inputLinesDevice will be the 2d array flattened 
    hipMemcpy((void*)inputLinesDevice, (void*)inputLine1d, lenInputLines * numInputLines * sizeof(char), hipMemcpyHostToDevice);
    char* patternLinesDevice;
    hipMalloc((void**)&patternLinesDevice, lenPatternLines * numPatternLines * sizeof(char));
    hipMemcpy((void*)patternLinesDevice, (void*)patternLine1d, lenPatternLines * numPatternLines * sizeof(char), hipMemcpyHostToDevice);

    //allocate memory for the length and num of lines for both files 
    int* numInputLinesDevice;
    hipMalloc((void**) & numInputLinesDevice, sizeof(int));
    hipMemcpy(numInputLinesDevice, & numInputLines, sizeof(int), hipMemcpyHostToDevice);
    int* lenInputLinesDevice;
    hipMalloc((void**) & lenInputLinesDevice, sizeof(int));
    hipMemcpy((void*)lenInputLinesDevice, &lenInputLines, sizeof(int), hipMemcpyHostToDevice);
    int* numPatternLinesDevice;
    hipMalloc((void**) & numPatternLinesDevice, sizeof(int));
    hipMemcpy((void*)numPatternLinesDevice, &numPatternLines, sizeof(int), hipMemcpyHostToDevice);
    int* lenPatternLinesDevice;
    hipMalloc((void**) & lenPatternLinesDevice, sizeof(int));
    hipMemcpy((void*)lenPatternLinesDevice, &lenPatternLines, sizeof(int), hipMemcpyHostToDevice);

    // set the number of blocks and number of threads we want to use
    int numBlocks = 1;
    int numThreads = 10;
    // guessing how to compute this... - maybe for smaller test files we can change this around but for larger it needs to be set
    // if (numInputLines <= 1024) {
    //     numThreads = numInputLines; // according to the internet, 1024 is the max number of threads in a block
    // } else {
    //     numBlocks = ; // ???
    //     numThreads = ; // ???
    // }
    int totalNumThreads = numBlocks * numThreads;

    // setup pointers to get the results from device memory in allMatchLocations and numMatchesArr
    matchLocation* allMatchLocationsDevice;
    hipMalloc(&allMatchLocationsDevice, 9999 * sizeof(matchLocation)); // temp.. limiting to 9999 partial matches
    int* numMatchesArrDevice;
    hipMalloc(&numMatchesArrDevice, sizeof(int));
    hipMemset(numMatchesArrDevice, 0, 1); // initialize numMatches to 0

    // start the kernel to find partial matches
    findPartialMatches<<<numBlocks,numThreads>>>(inputLinesDevice, patternLinesDevice, numInputLinesDevice, lenInputLinesDevice, numPatternLinesDevice, lenPatternLinesDevice, numMatchesArrDevice, allMatchLocationsDevice);
    cout << "after kernel exec (not necessarily done)" << endl;
    hipDeviceSynchronize();
    cout << "after sync" << endl;
    hipError_t err = hipGetLastError();
    cout << "got last err" << endl;
    cout << err << endl;
    // copy the results to host memory
    int numMatches;
    cout << " about to memcpy" << endl;
    hipMemcpy(&numMatches, numMatchesArrDevice, sizeof(int), hipMemcpyDeviceToHost);
    cout << "num mathces is " << numMatches << endl;

    matchLocation* allMatchLocations = new matchLocation[numMatches];
    hipMemcpy(allMatchLocations, allMatchLocationsDevice, numMatches * sizeof(matchLocation), hipMemcpyDeviceToHost);


    // prep the output file 
    ofstream outputFile("output.txt");

    // compare all the matches for lines in a pattern file to find full matches
    int sizeOfCoordArr = 10; // 10 for now... then dynamically increase if needed
    matchCoordinate* coordArr = new matchCoordinate[sizeOfCoordArr];
    int numCoords = 0;
    for (int i = 0; i < numMatches; i++) {
        matchCoordinate coor = searchForRealMatches(allMatchLocations[i], allMatchLocations, numMatches, numPatternLines, totalNumThreads);
        if (coor.x == -1 && coor.y == -1) continue; // not a match
        bool alreadyFound = false;
        for (int k = 0; k < numCoords; k++) {
            if (coordArr[k].x == coor.x && coordArr[k].y == coor.y) alreadyFound = true;
        }
        if (alreadyFound) continue; // go to next match
        if (numCoords >= sizeOfCoordArr) {
            // increase coordArr size 
            int biggerSize = sizeOfCoordArr * 2;
            matchCoordinate* biggerArr = new matchCoordinate[biggerSize];
            memcpy(biggerArr, coordArr, sizeof(matchCoordinate) * numCoords);
            delete[] coordArr;
            coordArr = biggerArr;
            sizeOfCoordArr = biggerSize;
        }
        coordArr[numCoords] = coor;
        numCoords++;
        cout << "MATCH AT: " << coor.x << ", " << coor.y << endl;
        // output as column, row and ensure the coordinates are not 0 indexed
        outputFile << coor.y + 1 << ", " << coor.x + 1 << "\n";
    }
    outputFile.close();
    // cleanup memory 
    for (int i = 0; i < numInputLines; i++) {
//        hipFree(inputLinesDevice[i]);
        delete[] inputLines[i];
    }
    delete[] inputLines;
    hipFree(inputLinesDevice);
    for (int i = 0; i < numPatternLines; i++) {
 //       hipFree(patternLinesDevice[i]);
        delete[] patternLines[i];
    }
    delete[] patternLines;
    hipFree(patternLinesDevice);
  //  for (int i = 0; i < totalNumThreads; i++) {
   //     hipFree(allMatchLocationsDevice[i]);
   ///     delete[] allMatchLocations[i];
   // }
    delete[] allMatchLocations;
    hipFree(allMatchLocationsDevice);
   // delete[] numMatchesArr;
    hipFree(numMatchesArrDevice);
    delete[] coordArr;

    return 0;
}
