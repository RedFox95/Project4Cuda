#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <string>

using namespace std;

struct fileinfo {
    int numLines;
    size_t lineLength;
};

struct matchLocation {
    int x; // row 
    int y; // col
    int pl; // pattern line
};

struct matchCoordinate {
    int x;
    int y;
};


//Get the fileinfo struct for this file, containing the number of lines and length of a line.

fileinfo getFileInfo(string filename) {
    ifstream lineCounter(filename);
    int numLines = 0;
    string line;
    while (getline(lineCounter, line)) numLines++;
    lineCounter.close();
    struct fileinfo retVal = { numLines, line.length() };
    return retVal;
}


//Returns the upper leftmost coordinate of a full match, otherwise return null if no full match.

matchCoordinate searchForRealMatches(matchLocation match, matchLocation** allMatches, int* numMatchesArr, int numPatternLines, int totalNumThreads) {
    //cout << "-> searchForRealMAtches x:" << match.x << " y: " << match.y << " pl: " << match.pl << endl;
    matchLocation** patternMatchLocations = new matchLocation * [numPatternLines];
    for (int i = 0; i < numPatternLines; i++) patternMatchLocations[i] = nullptr;

    patternMatchLocations[match.pl] = &match;
    for (int i = 0; i < totalNumThreads; i++) {
        for (int j = 0; j < numMatchesArr[i]; j++) {
            bool fullMatch = true;
            // for each match...
            if (allMatches[i][j].y == match.y && allMatches[i][j].x != match.x && allMatches[i][j].pl != match.pl) {
                // if it's in the correct column
                for (int k = 0; k < numPatternLines; k++) {
                    if (allMatches[i][j].x == match.x + k && allMatches[i][j].pl == match.pl + k) {
                        // this is a corresponding match!
                        patternMatchLocations[allMatches[i][j].pl] = &allMatches[i][j];
                    }
                    // check if full match 
                    if (patternMatchLocations[k] == nullptr) fullMatch = false;
                }
                if (fullMatch) {
                    struct matchCoordinate retVal = { patternMatchLocations[0]->x, patternMatchLocations[0]->y };
                    delete[] patternMatchLocations;
                    return retVal;
                }
            }
        }
    }
    // return -1, -1 if no match found 
    struct matchCoordinate retVal = { -1, -1 };
    return retVal;
}

__device__ void findSubStr(char*str, int row, int strLen, char* subStr, int patternLineNum, int subStrLen, int pos, int*foundPos) {
    printf("-> findSubStr row %d, strLen %d, patternLineNum %d, subStrLen %d, pos %d\n", row, strLen, patternLineNum, subStrLen, pos);
    //*foundPos = -1; // initialize to -1 to assume not found // TODO this causes memory problem - maybe need some sort of malloc type thing here
    //printf("r %d: difference is %d \n", row, strLen-subStrLen);
    *foundPos = -1; // assume no match found
    for (int i = pos; i <= strLen - subStrLen; i++) {
        printf("r %d: i is %d\n", row, i);
        bool found = true;
        for (int j = 0; j < subStrLen; j++) {
            printf("r %d: j is %d and i+j is %d comparing %c vs %c\n", row, j, i + j, str[(row * strLen) + i + j], subStr[(patternLineNum * subStrLen)  + j]);
            if (str[(row*strLen) + i + j] != subStr[(patternLineNum * subStrLen)+j]) {
                printf("r %d: not a match\n", row);
                found = false;
                break;
            }
            else {
                printf("r %d: part of a match!\n", row);
            }
        }
        if (found) {
            *foundPos = i;
            printf("r %d: FOUND match in device foundPos is %d\n", row, *foundPos);
            break;
        }
    }
}

__global__ void findPartialMatches(char* inputLines, char* patternLines, int* numInputLines, int* lenInputLines, int* numPatternLines, int* lenPatternLines, int* numMatchesArr, matchLocation** allMatches, int*sizeArr) {
//    printf("HELLO FROM KERNEL!\n");
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int threadIndex = threadId * *lenInputLines;
    printf("tmg numinputlines %d, numpattern lines %d\n", *numInputLines, *numPatternLines); 
    printf("tMg leninputlines %d, lenpattern lines %d\n", *lenInputLines, *lenPatternLines); 
    printf("Tmg thread id is %d and index is %d\n", threadId, threadIndex);
    numMatchesArr[threadId] = 0; // initialize all numMatches to 0
    sizeArr[threadId] = 10; // initialize all array sizes to 10
    printf("tId %d: nummatches set to %d and size set to %d\n", threadId, numMatchesArr[threadId], sizeArr[threadId]);
    
    allMatches[threadId] = new matchLocation[sizeArr[threadId]];
    if (threadId < *numInputLines) {
        //printf("de did numinputlines\n");
       // int sizeOfMatchArr = 10; // 10 for now... then dynamically increase if needed
        //allMatches[threadId] = new matchLocation[sizeOfMatchArr]; // store the matches at the index for this thread
        for (int j = 0; j < *numPatternLines; j++) {
            printf("tId %d: in loop j is %d\n", threadId, j);
            int pos = 0;
            //string jPatternLine(patternLines[j], patternLines[j] + *lenPatternLines - 1);
            //string iInputLine(inputLines[threadId], inputLines[threadId] + *lenInputLines - 1);
            //int found = iInputLine.find(jPatternLine, pos);
            int found = -1;
//            findSubStr(inputLines[threadId], *lenInputLines - 1, patternLines[j], *lenPatternLines - 1, 0, found);
            printf("before calling findSubStr j is %d\n", j);
            findSubStr(inputLines, threadId, *lenInputLines, patternLines, j, *lenPatternLines, 0, &found); // idk if these -1 are needed in this implementation
            printf("tId %d: after find found is %d and j is %d\n", threadId, found, j);
            while (found != -1) {
                if (numMatchesArr[threadId] >= sizeArr[threadId]) {
                    // increase matchArr size 
                    size_t biggerSize = sizeArr[threadId] * 2;
                    matchLocation* biggerArr = new matchLocation[biggerSize];
                    memcpy(biggerArr, allMatches[threadId], sizeof(matchLocation) * numMatchesArr[threadId]);
                    delete[] allMatches[threadId];
                    allMatches[threadId] = biggerArr;
                    sizeArr[threadId] = biggerSize;
                }
                // store the match
                struct matchLocation m = { threadId, found, j }; // where threadId is the row number, found is the col number, and j is the pattern line number
                printf("tID %d: made the matchLocation\n", threadId);
                printf("tID %d: allmatches arr storing in [%d][%d]\n", threadId, threadId, numMatchesArr[threadId]);
                allMatches[threadId][numMatchesArr[threadId]] = m;
                printf("tId %d: after storing matchlocation and before incrementing num matches which is currenmtly %d\n", threadId, numMatchesArr[threadId]);
                // update pos, numMatches, and found for the next iteration
                pos = found + 1;
                numMatchesArr[threadId]++;
                printf("tId %d: after incrementing nummatchesarr to %d\n", threadId, numMatchesArr[threadId]);
  //              findSubStr(inputLines[threadId], *lenInputLines - 1, patternLines[j], *lenPatternLines - 1, pos, found);
                printf("tId %d: abvout to call findSubStr again j is %d\n", threadId, j);
                findSubStr(inputLines, threadId,  *lenInputLines, patternLines, j, *lenPatternLines, pos, &found);
            }
            printf("tId %d: after while loop\n", threadId);
        }
        printf("tId %d: after for loop\n", threadId);
    }
    printf("Thread %d num matches is %d\n", threadId, numMatchesArr[threadId]);
}

int main(int argc, char** argv) {
    // get info about input file
    string inputFile = argv[1];
    fileinfo inputInfo = getFileInfo(inputFile);
    int numInputLines = inputInfo.numLines;
    int lenInputLines = inputInfo.lineLength;

    // read the input file in line by line and store in array
    ifstream file(inputFile);
    char** inputLines = new char* [numInputLines]; // num rows (lines)
    for (int i = 0; i < numInputLines; i++) {
        inputLines[i] = new char[lenInputLines+1]; // num cols (line length)
    }

    // temp test..
    char* inputLine1d = new char[numInputLines * lenInputLines];
    cout << " length of 1d arr is " << numInputLines * lenInputLines  << endl;

    string line;
    int lineNum = 0; // for indexing into the allLines arr
    while (getline(file, line)) {
        strcpy_s(inputLines[lineNum], lenInputLines+1, line.c_str());
        cout << inputLines[lineNum] << endl;
        for (int i = 0; i < lenInputLines; i++) {
            cout << "inserting " << line.c_str()[i] << " at " << (lineNum * lenInputLines) + i << endl;
            inputLine1d[(lineNum * lenInputLines) + i] = line.c_str()[i];
        }
        
        lineNum++;
    }
    for (int i = 0; i < numInputLines * lenInputLines; i++) {
        cout << inputLine1d[i];
    }
    cout << "\nthat was it " << endl;
    // get info about the pattern file
    string patternFile = argv[2];
    fileinfo patternInfo = getFileInfo(patternFile);
    int numPatternLines = patternInfo.numLines;
    int lenPatternLines = patternInfo.lineLength;

    // read the pattern file in line by line and store in array
    ifstream patternFileStream(patternFile);
    char** patternLines = new char* [numPatternLines]; // num rows (lines)
    for (int i = 0; i < numPatternLines; i++) {
        patternLines[i] = new char[lenPatternLines+1]; // num cols (line length)
    }
    char* patternLine1d = new char[numPatternLines*lenPatternLines];
    cout << " length of 1d arr is " << numPatternLines * lenPatternLines << endl;

    lineNum = 0; // for indexing into the pattern arr
    while (getline(patternFileStream, line)) {
        strcpy_s(patternLines[lineNum], lenPatternLines+1, line.c_str());
        for (int i = 0; i < lenPatternLines; i++) {
            cout << "inserting " << line.c_str()[i] << " at " << (lineNum * lenPatternLines) + i << endl;
            patternLine1d[(lineNum * lenPatternLines) + i] = line.c_str()[i];
        }
        lineNum++;
    }
    for (int i = 0; i < numPatternLines * lenPatternLines; i++) {
        cout << patternLine1d[i];
    }
    cout << "\nthat was it " << endl;
    cout << "numinput lines is " << numInputLines << " and num pattern lines is " << numPatternLines << endl;
    cout << "about to allocate mem for inputLines and patternlines" << endl;
    // allocate memory on device for inputLines and patternLines and copy to the device memory
    char* inputLinesDevice;
  //  hipMalloc((void**)&inputLinesDevice, lenInputLines * numInputLines * sizeof(char)); // inputLinesDevice will be the 2d array flattened 
    hipMalloc((void**)&inputLinesDevice, lenInputLines * numInputLines * sizeof(char)); // inputLinesDevice will be the 2d array flattened 
    cout << "cudamalloc for inputLinesDevice done" << endl;
  //  hipMemcpy((void*)inputLinesDevice, (void*)inputLines, lenInputLines * numInputLines * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy((void*)inputLinesDevice, (void*)inputLine1d, lenInputLines * numInputLines * sizeof(char), hipMemcpyHostToDevice);
    cout << "hipMemcpy for inputLinesDevice done" << endl;
  /*  for (int i = 0; i < numInputLines; i++) {
        cout << " in loo i is " << i << endl;
        hipMalloc((void**) & (inputLinesDevice[i]), lenInputLines * sizeof(char));
        cout << " malloc not to memcpy" << endl;
        hipMemcpy(inputLinesDevice[i], inputLines[i], lenInputLines * sizeof(char), hipMemcpyHostToDevice);
    }*/
   // size_t inputPitch;
    //hipMallocPitch((void**)inputLinesDevice, &inputPitch,  lenInputLines * sizeof(char), numInputLines);
    //size_t spitch = numPatternLines * lenInputLines *sizeof(char);
    //hipMemcpy2DToArray((void**)inputLinesDevice, 0, 0, inputLines, spitch, lenPatternLines * sizeof(char), numInputLines, hipMemcpyHostToDevice);
    cout << "malloc and memcpy done for each line of input" << endl;
    char* patternLinesDevice;
    cout << "about to malloc pattern" << endl;
//    hipMalloc((void**)&patternLinesDevice, lenPatternLines * numPatternLines * sizeof(char));
    hipMalloc((void**)&patternLinesDevice, lenPatternLines * numPatternLines * sizeof(char));
    cout << "after malloc 1" << endl;
//    hipMemcpy((void*)patternLinesDevice, (void*)patternLines, lenPatternLines * numPatternLines * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy((void*)patternLinesDevice, (void*)patternLine1d, lenPatternLines * numPatternLines * sizeof(char), hipMemcpyHostToDevice);
    cout << "after memcpy1" << endl;
   /* for (int i = 0; i < numPatternLines; i++) {
        hipMalloc(&patternLinesDevice[i], lenPatternLines * sizeof(char));
        hipMemcpy(patternLinesDevice[i], patternLines[i], lenPatternLines * sizeof(char), hipMemcpyHostToDevice);
    }
    cout << "same done for pattern" << endl;
    */
    //allocate memory for the length and num of lines for both files 
    int* numInputLinesDevice;
    hipMalloc((void**) & numInputLinesDevice, sizeof(int));
    hipMemcpy(numInputLinesDevice, & numInputLines, sizeof(int), hipMemcpyHostToDevice);
    int* lenInputLinesDevice;
    hipMalloc((void**) & lenInputLinesDevice, sizeof(int));
    hipMemcpy((void*)lenInputLinesDevice, &lenInputLines, sizeof(int), hipMemcpyHostToDevice);
    int* numPatternLinesDevice;
    hipMalloc((void**) & numPatternLinesDevice, sizeof(int));
    hipMemcpy((void*)numPatternLinesDevice, &numPatternLines, sizeof(int), hipMemcpyHostToDevice);
    int* lenPatternLinesDevice;
    hipMalloc((void**) & lenPatternLinesDevice, sizeof(int));
    hipMemcpy((void*)lenPatternLinesDevice, &lenPatternLines, sizeof(int), hipMemcpyHostToDevice);

    // set the number of blocks and number of threads we want to use
    int numBlocks = 1;
    int numThreads = 10;
    // guessing how to compute this... - maybe for smaller test files we can change this around but for larger it needs to be set
    // if (numInputLines <= 1024) {
    //     numThreads = numInputLines; // according to the internet, 1024 is the max number of threads in a block
    // } else {
    //     numBlocks = ; // ???
    //     numThreads = ; // ???
    // }
    int totalNumThreads = numBlocks * numThreads;

//    int* numThreadsDevice;
//    hipMalloc((void**)&numThreadsDevice, sizeof(int));
//    hipMemcpy((void*)numThreadsDevice, &numThreads, sizeof(int), hipMemcpyHostToDevice);

    // setup pointers to get the results from device memory in allMatchLocations and numMatchesArr
    matchLocation** allMatchLocationsDevice;
    hipMalloc(&allMatchLocationsDevice, totalNumThreads * sizeof(matchLocation*));
    cout << "after malloc mathclocations device " << endl;
    int* numMatchesArrDevice;
    hipMalloc(&numMatchesArrDevice, totalNumThreads * sizeof(int));
    int* sizeArrDevice;
    hipMalloc(&sizeArrDevice, totalNumThreads * sizeof(int));
//    cout << "after malloc num matches device " << endl; // idk if this is needed
//    for (int i = 0; i < numBlocks; i++) {
//        cout << i << endl;
//        hipMalloc((void**) &allMatchLocationsDevice[i], numBlocks * sizeof(int));
//    }
    cout << "about to start kernel" << endl;
    // start the kernel to find partial matches
    findPartialMatches<<<numBlocks,numThreads>>>(inputLinesDevice, patternLinesDevice, numInputLinesDevice, lenInputLinesDevice, numPatternLinesDevice, lenPatternLinesDevice, numMatchesArrDevice, allMatchLocationsDevice, sizeArrDevice);
    cout << "after kernel exec (not necessarily done)" << endl;
    hipDeviceSynchronize();
    cout << "after sync" << endl;
    hipError_t err = hipGetLastError();
    cout << "got last err" << endl;
    cout << err << endl;
    // copy the results to host memory
    int* numMatchesArr = new int[totalNumThreads];
    matchLocation** allMatchLocations = new matchLocation * [totalNumThreads];

    for (int i = 0; i < totalNumThreads; i++) {
        hipMemcpy((void*)numMatchesArr[i], (void*) & numMatchesArrDevice[i], sizeof(int), hipMemcpyDeviceToHost);
        cout << "nummathces is " << numMatchesArr[i] << endl;
        if (numMatchesArr[i] > 0) {
            hipMemcpy(allMatchLocations[i], allMatchLocationsDevice[i], numMatchesArr[i] * sizeof(matchLocation*), hipMemcpyDeviceToHost);
        }
    }
    cout << "after copying mem from dev to host for nummatch which is " << numMatchesArr[1] << " and allmatchloc" << endl;

    // prep the output file 
    ofstream outputFile("output.txt");

    // compare all the matches for lines in a pattern file to find full matches
    int sizeOfCoordArr = 10; // 10 for now... then dynamically increase if needed
    matchCoordinate* coordArr = new matchCoordinate[sizeOfCoordArr];
    int numCoords = 0;
    for (int i = 0; i < numBlocks; i++) {
        for (int j = 0; j < numMatchesArr[i]; j++) {
            matchCoordinate coor = searchForRealMatches(allMatchLocations[i][j], allMatchLocations, numMatchesArr, numPatternLines, totalNumThreads);
            if (coor.x == -1 && coor.y == -1) continue; // not a match
            bool alreadyFound = false;
            for (int k = 0; k < numCoords; k++) {
                if (coordArr[k].x == coor.x && coordArr[k].y == coor.y) alreadyFound = true;
            }
            if (alreadyFound) continue; // go to next match
            if (numCoords >= sizeOfCoordArr) {
                // increase coordArr size 
                int biggerSize = sizeOfCoordArr * 2;
                matchCoordinate* biggerArr = new matchCoordinate[biggerSize];
                memcpy(biggerArr, coordArr, sizeof(matchCoordinate) * numCoords);
                delete[] coordArr;
                coordArr = biggerArr;
                sizeOfCoordArr = biggerSize;
            }
            coordArr[numCoords] = coor;
            numCoords++;
            cout << "MATCH AT: " << coor.x << ", " << coor.y << endl;
            // output as column, row and ensure the coordinates are not 0 indexed
            outputFile << coor.y + 1 << ", " << coor.x + 1 << "\n";
        }
    }
    outputFile.close();
    // cleanup memory 
    for (int i = 0; i < numInputLines; i++) {
//        hipFree(inputLinesDevice[i]);
        delete[] inputLines[i];
    }
    delete[] inputLines;
    hipFree(inputLinesDevice);
    for (int i = 0; i < numPatternLines; i++) {
 //       hipFree(patternLinesDevice[i]);
        delete[] patternLines[i];
    }
    delete[] patternLines;
    hipFree(patternLinesDevice);
    for (int i = 0; i < totalNumThreads; i++) {
        hipFree(allMatchLocationsDevice[i]);
        delete[] allMatchLocations[i];
    }
    delete[] allMatchLocations;
    hipFree(allMatchLocationsDevice);
    delete[] numMatchesArr;
    hipFree(numMatchesArrDevice);
    delete[] coordArr;

    return 0;
}

